#include "hip/hip_runtime.h"
#include <stdint.h>
#include <hip/hip_runtime.h>
#include "utils.h"

__global__
void convert_from_tiled(int* d_table, const bboard* d_a, const size_t dim, const size_t dim_board, const size_t pitch) {
    UNUSED(dim_board);

    // coordinates in the tiled table.
    const int major_i = blockIdx.y * blockDim.y + threadIdx.y;
    const int major_j = blockIdx.x * blockDim.x + threadIdx.x;
    if (major_i * WIDTH >= dim) return;
    if (major_j * WIDTH >= dim) return;

    const bboard* row_a = (bboard*)((char*)d_a + major_j * pitch);
    const bboard value = row_a[major_i];

    for (int board_i = 0; board_i < WIDTH; board_i++){
        const int real_i = major_i * WIDTH + board_i;
        if (real_i >= dim) break;
        for (int board_j = 0; board_j < WIDTH; board_j++){
            const int real_j = major_j * WIDTH + board_j;
            if (real_j >= dim) break;
            const int real_idx = real_i * dim + real_j;
            d_table[real_idx] = BOARD_IS_SET(value, board_i, board_j);
        }
    }
}

__global__
void convert_to_tiled(const int* d_table, bboard* d_a, const size_t dim, const size_t dim_board, const size_t pitch) {
    UNUSED(dim_board);

    const int major_i = blockIdx.y * blockDim.y + threadIdx.y;
    const int major_j = blockIdx.x * blockDim.x + threadIdx.x;
    if (major_i * WIDTH >= dim) return;
    if (major_j * WIDTH >= dim) return;


    bboard* row_a = (bboard*)((char*)d_a + major_j * pitch);
    bboard value = 0;

    for (int board_i = 0; board_i < WIDTH; board_i++){
        const int real_i = major_i * WIDTH + board_i;
        if (real_i >= dim) break;
        for (int board_j = 0; board_j < WIDTH; board_j++){
            const int real_j = major_j * WIDTH + board_j;
            if (real_j >= dim) break;
            const int real_idx = real_i * dim + real_j;
            if (d_table[real_idx]) {
                SET_BOARD(value, board_i, board_j);
            }
        }
    }
    row_a[major_i] = value;
}
