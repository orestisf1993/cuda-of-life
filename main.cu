#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "utils.h"

static inline void cudaCheckErrors(const char msg[], const char file[], int line) {
    do {
        hipError_t __err = hipGetLastError();

        if (__err != hipSuccess) {
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n",
                    msg, hipGetErrorString(__err),
                    file, line);
            hipDeviceReset();
            exit(1);
        }
    } while (0);
}

#define POS(i,j) (i + WIDTH * j)

#define SET_BIT(val, bit_idx) val |= (ONE << bit_idx)
#define SET_BOARD(val, i, j) SET_BIT(val, POS(i, j))
#define CLEAR_BIT(val, bit_idx) val &= ~(ONE << bit_idx)
#define CLEAR_BOARD(val, i, j) CLEAR_BIT(val, POS(i, j))
#define TOGGLE_BIT(val, bit_idx) val ^= (ONE << bit_idx)
#define TOGGLE_BOARD(val, i, j) TOGGLE_BIT(val, POS(i, j))
#define BIT_IS_SET(val, bit_idx) (val & (ONE << bit_idx))
#define BOARD_IS_SET(val, i, j) BIT_IS_SET(val, POS(i, j))

__global__
void convert_to_tiled(int* d_table, bboard* d_board, size_t dim, size_t dim_board) {
    int major_i = blockIdx.y * blockDim.y + threadIdx.y;
    int major_j = blockIdx.x * blockDim.x + threadIdx.x;
    int board_i = 0;
    int board_j = 0;
    int real_i = major_i * WIDTH + board_i;
    int real_j = major_j * WIDTH + board_j;
//    int idx = row * WIDTH + col;
}

int main(int argc, char** argv) {
    if (argc < 6) {
        printf("usage: %s fname dim (iter blockx blocky gridx gridy)\n", argv[0]);
        exit(1);
    }
    int n_runs = DFL_RUNS;
    if (argc >= 4) {
        n_runs = atoi(argv[3]);
    }
    const size_t dim = atoi(argv[2]);
    const size_t total_elements = dim * dim;
    const size_t mem_size = total_elements * sizeof(int);
    dim3 block;
    dim3 grid;

    if (argc >= 6) {
        block.x = atoi(argv[4]);
        block.y = block.x;
        grid.x = atoi(argv[6]);
        grid.y = grid.x;
    }

    char* filename = argv[1];
    int* table;
    table = (int*) malloc(mem_size);
    read_from_file(table, filename, dim);

    bboard* d_board;
    int* d_table;
    const size_t dim_board = CEIL_DIV(dim, WIDTH);
    const size_t mem_size_board = dim_board * dim_board * sizeof(bboard);
    hipMalloc((void**) &d_table,  mem_size);
    cudaCheckErrors("device allocation of GOL matrix failed", __FILE__, __LINE__);
    hipMalloc((void**) &d_board, mem_size_board);
    cudaCheckErrors("device allocation of GOL tiled matrix failed", __FILE__, __LINE__);

    bboard* devPtr;
    size_t pitch;
    hipMallocPitch((void**)&devPtr, &pitch, dim_board * sizeof(bboard), dim_board);

    hipMemcpy(d_table, table, mem_size, hipMemcpyHostToDevice);
    cudaCheckErrors("copy from host to device memory failed", __FILE__, __LINE__);

    convert_to_tiled <<< grid, block >>> (d_table, d_board, dim, dim_board);

    return 0;
}

