#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "utils.h"

#define T_I 0
#define C_I 1
#define B_I 2
#define L_J 0
#define C_J 1
#define R_J 2

__global__ void calculate_next_generation(const bboard* d_a, bboard* d_result, const int dim,
    const int dim_board, const size_t pitch) {
  const int major_i = blockIdx.y * blockDim.y + threadIdx.y;  // row
  const int major_j = blockIdx.x * blockDim.x + threadIdx.x;  // col
  if (major_i * WIDTH >= dim) return;
  if (major_j * WIDTH >= dim) return;

  const int major_l = (major_j - 1 + dim_board) % dim_board;
  const int major_r = (major_j + 1) % dim_board;
  const int major_t = (major_i - 1 + dim_board) % dim_board;
  const int major_b = (major_i + 1) % dim_board;

  bboard* row_c = (bboard*)((char*)d_a + major_i * pitch);
  bboard* row_t = (bboard*)((char*)d_a + major_t* pitch);
  bboard* row_b = (bboard*)((char*)d_a + major_b * pitch);

  bboard neighbors[3][3];
  neighbors[C_I][C_J] = row_c[major_j];
  neighbors[C_I][L_J] = row_c[major_l];
  neighbors[C_I][R_J] = row_c[major_r];
  neighbors[T_I][C_J] = row_t[major_j];
  neighbors[T_I][L_J] = row_t[major_l];
  neighbors[T_I][R_J] = row_t[major_r];
  neighbors[B_I][C_J] = row_b[major_j];
  neighbors[B_I][L_J] = row_b[major_l];
  neighbors[B_I][R_J] = row_b[major_r];

  //TODO: move global?
  const int remaining_dim = gridDim.x * blockDim.x * WIDTH - dim;
  //    const int remaining_blocks = remaining_dim / WIDTH; // for shared??
  const int remaining_cells = remaining_dim % WIDTH;
  //    assert(remaining_dim >= 0);
  //    assert(remaining_blocks >= 0);
  //    assert(remaining_cells >= 0);

  const bool is_edge_r = (major_j == dim_board - 1);
  const bool is_edge_d = (major_i == dim_board - 1);
  const bool is_edge_u = (major_i == 0);
  const bool is_edge_l = (major_j == 0);

  const int limit_i = WIDTH - remaining_cells * is_edge_d;
  const int limit_j = WIDTH - remaining_cells * is_edge_r;

  if (is_edge_d) CUDA_PRINT("%d %d is_edge_d\n", major_i, major_j);
  if (is_edge_r) CUDA_PRINT("%d %d is_edge_r\n", major_i, major_j);
  if (is_edge_u) CUDA_PRINT("%d %d is_edge_u\n", major_i, major_j);
  if (is_edge_l) CUDA_PRINT("%d %d is_edge_l\n", major_i, major_j);

  bboard value = 0;
  int up_i, up_n, down_i, down_n;
  int right_j, right_n;
  uint first_cells, second_cells;
  uint alive_cells, this_cell;
  for (int i = 0; i < limit_i; i++) {

    if (i == 0) {
      up_i = WIDTH - 1 - remaining_cells * is_edge_u;
      up_n = T_I;
    } else {
      up_i = i - 1;
      up_n = C_I;
    }
    if (i == limit_i - 1) {
      down_i = 0;
      down_n = B_I;
    } else {
      down_i = i + 1;
      down_n = C_I;
    }

    for (int j = 0; j < limit_j; j++) {

      this_cell = BOARD_IS_SET(neighbors[C_I][C_J], i, j);

      if (j == 0) {
        int left_j = WIDTH - 1 - remaining_cells * is_edge_l;
        first_cells = BOARD_IS_SET(neighbors[up_n][L_J], up_i, left_j)
          + BOARD_IS_SET(neighbors[C_I][L_J], i, left_j)
          + BOARD_IS_SET(neighbors[down_n][L_J], down_i, left_j);
        second_cells = BOARD_IS_SET(neighbors[up_n][C_J], up_i, j)
          + this_cell
          + BOARD_IS_SET(neighbors[down_n][C_J], down_i, j);
      }
      if (j == limit_j - 1) {
        right_j = 0;
        right_n = R_J;
      } else {
        right_j = j + 1;
        right_n = C_J;
      }

      if (j & 1u) {
        alive_cells = second_cells;
        second_cells = BOARD_IS_SET(neighbors[up_n][right_n], up_i, right_j)
          + BOARD_IS_SET(neighbors[C_I][right_n], i, right_j)
          + BOARD_IS_SET(neighbors[down_n][right_n], down_i, right_j);
        alive_cells += second_cells;
        alive_cells += first_cells - this_cell;
      }
      else {
        alive_cells = first_cells;
        first_cells = BOARD_IS_SET(neighbors[up_n][right_n], up_i, right_j)
          + BOARD_IS_SET(neighbors[C_I][right_n], i, right_j)
          + BOARD_IS_SET(neighbors[down_n][right_n], down_i, right_j);
        alive_cells += first_cells;
        alive_cells += second_cells - this_cell;
      }

      const bool set = (alive_cells == 3) || (alive_cells == 2 && this_cell);
      if (set) {
        SET_BOARD(value, i, j);
      }
    }

  }

  bboard* row_result = (bboard*)((char*)d_result + major_i * pitch);
  row_result[major_j] = value;
}
